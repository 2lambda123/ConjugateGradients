/*Contains implementation for gpu_cg_solver functions.*/


#include "cg_colver_gpu.h"
#include "ckernels.h"

extern "C"
{
#include "gpu_utils.h"
#include "utils.h"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#define threadsPerBlock 256

#define FREE_DEVICE_STACK \
	hipFree(d_r);\
	hipFree(d_helper);\
	hipFree(d_x);\
	hipFree(d_rhs);\
	hipFree(d_d);\
	hipFree(d_Ax);\
	hipFree(d_q);\
	hipFree(d_val);\
	hipFree(d_I);\
	hipFree(d_J);\
	hipFree(d_beta);\
	hipFree(d_alfa);\
	hipFree(d_alpha_zero);\
	hipFree(d_dot);\
	hipFree(d_norm);\
	hipFree(d_dot_zero);\
	hipFree(d_dot_old);\
	hipFree(d_dTq);


int gpu_conjugate_gradient_solver(Matrix *matrix, double *x_vec, double *rhs, double *res_vec, GPU_data gpu_data){
	/*Single GPU CG solver using cublas*/

	double *h_dot, *h_dot_zero;
	int *d_I = NULL, *d_J = NULL;
	const double tol = 1e-2f;
	double *d_alfa, *d_beta, *d_alpha_zero;
	double *d_Ax, *d_x, *d_d, *d_q, *d_rhs, *d_r, *d_helper, *d_norm, *d_dot, *d_dot_zero, *d_dot_old, *d_dTq, *d_val;
	int k, max_iter;

	k = 0;
	h_dot = 0;
	h_dot_zero = 0;
	max_iter = 200;

	size_t size = matrix->size * sizeof(double);

	hipsparseHandle_t cusparseHandle = 0;
	hipblasHandle_t cublasHandle = 0;
	hipsparseMatDescr_t descr = 0;

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	hipMalloc((void **)&d_r, size);
	hipMalloc((void **)&d_helper, size);
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_rhs,	size);
	hipMalloc((void **)&d_d, size);
	hipMalloc((void **)&d_Ax, size);
	hipMalloc((void **)&d_q, size);

	hipMalloc((void **)&d_val, matrix->non_zero * sizeof(double));
	hipMalloc((void **)&d_J, matrix->non_zero * sizeof(double));
	hipMalloc((void **)&d_I, (matrix->size + 1) * sizeof(int));

	hipMalloc((void **)&d_beta, sizeof(double));
	hipMalloc((void **)&d_alfa, sizeof(double));
	hipMalloc((void **)&d_alpha_zero, sizeof(double));
	hipMalloc((void **)&d_dot, sizeof(double));
	hipMalloc((void **)&d_dot_zero, sizeof(double));
	hipMalloc((void **)&d_norm, sizeof(double));

	hipMemcpy(d_val, matrix->val, matrix->non_zero * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_J, matrix->J_row, matrix->non_zero * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_I, matrix->I_column, (matrix->size + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_x, x_vec, size, hipMemcpyHostToDevice);
	hipMemcpy(d_rhs, rhs, size, hipMemcpyHostToDevice);

	int blocksPerGrid = ((matrix->size + threadsPerBlock -1) / threadsPerBlock );
	while (blocksPerGrid % threadsPerBlock != 0){
		blocksPerGrid++;
	}
	double alpha = 1.0;
	double beta = 0.0;

	const double one = 1.0;
	const double minus_one = -1.0;
	/*Calculate Ax matrix*/
	hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->size, matrix->size, matrix->non_zero,
					  &alpha, descr, d_val, d_J, d_I, d_x, &beta, d_Ax);
	/*Calculate rhs=rhs-Ax matrix*/
	hipblasDaxpy(cublasHandle, matrix->size, &minus_one, d_Ax, 1, d_rhs, 1);
	/*CG: Copy updated rhs (residuum) to d vector*/
	hipblasDcopy(cublasHandle, matrix->size, d_d, 1, d_rhs, 1);
	/*CG: calculate dot r'*r, assign it to dot_new */
	hipblasDdot(cublasHandle, matrix->size, d_rhs, 1, d_rhs, 1, d_dot);
	/*assign dot_new to dot_zero*/
	d_dot_zero = d_dot;
	hipMemcpy(h_dot, d_dot,  sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_dot_zero, d_dot_zero,  sizeof(double), hipMemcpyDeviceToHost);
	while ((*h_dot >  tol * tol * *h_dot_zero) && (k < max_iter)) {
		/*Calculate q=A*d vector*/
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->size, matrix->size, matrix->non_zero,
						  &alpha, descr, d_val, d_J, d_I, d_x, &beta, d_Ax);
		/*Calculate alpha:*/
		hipblasDdot(cublasHandle, matrix->size, d_d, 1, d_q, 1, d_dTq);
		sDdiv<<<1, gpu_data.devices[0].warp_size>>>(d_alfa, d_dot, d_dTq);
		/*Calculate x=x+alpha*d*/
		hipblasDaxpy(cublasHandle, matrix->size, d_alfa, d_x, 1, d_d, 1);
		/*Calculate r=r-alpha*q*/
		axpy<<<blocksPerGrid, threadsPerBlock>>>(matrix->size, -1, d_q, d_rhs);
		/*Assign dot_old = dot_new*/
		hipblasDcopy(cublasHandle, 1, d_dot_old, 1, d_dot, 1);
		/*CG:Assign dot_new = r'*r*/
		hipblasDdot(cublasHandle, matrix->size, d_rhs, 1, d_rhs, 1, d_dot);
		sDdiv<<<1, gpu_data.devices[0].warp_size>>>(d_beta, d_dot, d_dot_old);
		/*Scale beta*d*/
		hipblasDscal(cublasHandle, matrix->size, d_beta, d_d, 1);
		/*CG:Calculate d=r+beta*d*/
		hipblasDaxpy(cublasHandle, matrix->size, &one, d_rhs, 1, d_d, 1);
		k++;
	}
	hipsparseDestroy(cusparseHandle);
	hipDeviceReset();
	FREE_DEVICE_STACK
	return k;
}
}