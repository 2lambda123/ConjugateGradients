#include "hip/hip_runtime.h"
/*Contains implementation of custom kernels for CUDA devices.*/

#include "ckernels.h"
#include <hipblas.h>

const char* cublasGetErrorString(hipblasStatus_t status)
{
	switch(status)
	{
		case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
		case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
		case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
		case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
		case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
		case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
		case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
		case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}
	return "unknown error";
}

__global__ void sDdiv(double *res, double *divided, double *divider) {
	/*Division of scalar elements on a single CUDA thread*/
	if (threadIdx.x == 0) {
		res[0] = divided[0] / divider[0];
	}
}

__global__ void axpy(int num_elements, double alpha, double *x, double *y) {
	/*Perform computations of AXPY operations: y[i] = y[i] + alpha * x[i]*/
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < num_elements) {
		y[i] = y[i] + alpha * x[i];
	}
}

